#include <stdlib.h>
#include <stdio.h>
#include </usr/local/cuda-6.5/targets/x86_64-linux/include/hipblas.h>

#define index(i,j,ld) (((j)*(ld))+(i))


#include "Cuda_Utilities.h"

namespace Cuda_Utilities
{

void printMat(float*P,int uWP,int uHP){
//printf("\n %f",P[1]);
int i,j;
for(i=0;i<uHP;i++){

    printf("\n");

    for(j=0;j<uWP;j++)
        printf("%lg ",P[index(i,j,uHP)]);
        //printf("%lg ",P[i*uWP+j]);
}
}

extern "C" void matrixMul(int HA, int WA, int HB, int WB, int HC, int WC,
                          int HRes, int WRes,  
                          float* A, float* B, float* C, float* Res)
{
    int HresAB = HA;
    int WresAB = WB;
    HRes = HresAB;
    WRes = WC;
    // Not sure it should be here!
    cublasInit();

    hipblasStatus_t status;
    float* AA; float* BB; float* CC; float* resAB; float* res;
    /*ALLOCATE ON THE DEVICE*/
    status=cublasAlloc(HA*WA,sizeof(float),(void**)&AA);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory allocation error (AA)\n");
    }

    status=cublasAlloc(HB*WB,sizeof(float),(void**)&BB);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory allocation error (BB)\n");
    }

    status=cublasAlloc(HC*WC,sizeof(float),(void**)&CC);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory allocation error (CC)\n");
    }


    status=cublasAlloc(HA*WB,sizeof(float),(void**)&resAB);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory allocation error (ResAB)\n");
    }

    status=cublasAlloc(HRes*WRes,sizeof(float),(void**)&res);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory allocation error (ResAB)\n");
    }

    /*SET MATRIX*/
    status=hipblasSetMatrix(HA,WA,sizeof(float),A,HA,AA,HA);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory copy error (A)\n");
    }

    status=hipblasSetMatrix(HB,WB,sizeof(float),B,HB,BB,HB);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory copy error (B)\n");
    }

    status=hipblasSetMatrix(HC,WC,sizeof(float),C,HC,CC,HC);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device memory copy error (C)\n");
    }

     /*for (int i=0;i<HA*WA;i++)
        fprintf(stderr, "%lg ", A[i]);
        fprintf(stderr,"\n");
    for (int i=0;i<HB*WB;i++)
        fprintf(stderr, "%lg ", B[i]);
     fprintf(stderr,"\n");
     float* checkA = (float*)malloc(HA*WA*sizeof(float));
     hipblasGetMatrix(HA,WA,sizeof(float),AA,HA,checkA,HA);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device read error (A)\n");
    }
     for (int i=0;i<HA*WA;i++)
        fprintf(stderr, "%lg ", checkA[i]);
        fprintf(stderr, "\n");

        float* checkB = (float*)malloc(HB*WB*sizeof(float));
     hipblasGetMatrix(HB,WB,sizeof(float),BB,HB,checkB,HB);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device read error (A)\n");
    }
    /* for (int i=0;i<HB*WB;i++)
        fprintf(stderr, "%lg ", checkB[i]);
        fprintf(stderr,"\n");

     fprintf (stderr, "%d %d %d %d %d %d", HA, WA, HB, WB, HC, WC);
    fprintf(stderr,"hihi\n");*/
    /*KERNEL*/
    hipblasSgemm('n','n',HA,WB,WA,1,AA,HA,BB,HB,0,resAB,HresAB);

    status = cublasGetError();
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! kernel execution error.\n");
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
            fprintf (stderr, "HIPBLAS_STATUS_NOT_INITIALIZED\n");
        else if (status == HIPBLAS_STATUS_INVALID_VALUE)
            fprintf(stderr, "HIPBLAS_STATUS_INVALID_VALUE\n");  
        else if (status == HIPBLAS_STATUS_ARCH_MISMATCH) 
            fprintf(stderr, "HIPBLAS_STATUS_ARCH_MISMATCH\n");
        else if (status == HIPBLAS_STATUS_EXECUTION_FAILED)
            fprintf (stderr, "HIPBLAS_STATUS_EXECUTION_FAILED\n");        
    }

    hipblasSgemm('n','n',HresAB,WC,WresAB,1,resAB,HresAB,CC,HC,0,res,HRes);

    status = cublasGetError();
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! kernel execution error.\n");
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED)
            fprintf (stderr, "HIPBLAS_STATUS_NOT_INITIALIZED\n");
        else if (status == HIPBLAS_STATUS_INVALID_VALUE)
            fprintf(stderr, "HIPBLAS_STATUS_INVALID_VALUE\n");  
        else if (status == HIPBLAS_STATUS_ARCH_MISMATCH) 
            fprintf(stderr, "HIPBLAS_STATUS_ARCH_MISMATCH\n");
        else if (status == HIPBLAS_STATUS_EXECUTION_FAILED)
            fprintf (stderr, "HIPBLAS_STATUS_EXECUTION_FAILED\n");        
    }

    hipblasGetMatrix(HA,WC,sizeof(float),res,HA,Res,HA);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! device read error (Res)\n");
    }
    /*for (int i=0;i<HRes*WRes;i++)
        fprintf(stderr, "%lg ", Res[i]);
        fprintf(stderr,"\n");*/

    
    status = cublasFree(AA);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        fprintf (stderr, "!!!! memory free error (AA)\n");
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! memory free error (A)\n");
        }
        status = cublasFree(BB);
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! memory free error (B)\n");
        }
        status = cublasFree(CC);
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! memory free error (C)\n");
        }
        status = cublasFree(resAB);
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! memory free error (resAB)\n");
        }
        status = cublasFree(res);
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! memory free error (res)\n");
        }

        /* Shutdown */
        status = cublasShutdown();
        if (status != HIPBLAS_STATUS_SUCCESS) 
        {
            fprintf (stderr, "!!!! shutdown error (A)\n");
        }
    }
}

}